
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.1415926535897932384626433832795f
#endif

extern "C" __global__ void gather_kernel(float2 *g, float2 *f, float *theta, 
                                         int m, float mu, 
                                         int center_size, int n, int nproj, int nz)    
{

  const int center_half_size = center_size/2;

  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;

  if (tx >= n || ty >= nproj || tz >= nz)
    return;
  float2 g0, g0t;
  float w, coeff0;
  float w0, w1, x0, y0, coeff1;
  int ell0, ell1, g_ind, f_ind;

  g_ind = tx + ty * n + tz * n * nproj;
  coeff0 = M_PI / mu;
  coeff1 = -M_PI * M_PI / mu;
  float sintheta, costheta;
  __sincosf(theta[ty], &sintheta, &costheta);
  x0 = (tx - n / 2) / (float)n * costheta;
  y0 = -(tx - n / 2) / (float)n * sintheta;
  if (x0 >= 0.5f)
    x0 = 0.5f - 1e-5;
  if (y0 >= 0.5f)
    y0 = 0.5f - 1e-5;

  int stride1 = 2*n + 2*m;
  int stride2 = stride1 * stride1;

  /*float radius_2 = float(center_half_size - m - 1) * float(center_half_size - m - 1) / stride2;
  if( radius_2 > x0 * x0 + y0 * y0 )
    return;*/

  g0.x = g[g_ind].x;
  g0.y = g[g_ind].y;
  // offset f by [tz, n+m, n+m]

  f += n+m + (n+m) * stride1 + tz * stride2;
  //f_center += tz * memory_multiplier * center_size * center_size;
  #pragma unroll
  for (int i1 = 0; i1 < 2 * m + 1; i1++)
  {
    ell1 = floorf(2 * n * y0) - m + i1;
    #pragma unroll
    for (int i0 = 0; i0 < 2 * m + 1; i0++)
    {
      ell0 = floorf(2 * n * x0) - m + i0;
      w0 = ell0 / (float)(2 * n) - x0;
      w1 = ell1 / (float)(2 * n) - y0;
      w = coeff0 * __expf(coeff1 * (w0 * w0 + w1 * w1));
      g0t.x = w*g0.x;
      g0t.y = w*g0.y;
      f_ind = ell0 + stride1 * ell1 ;
      //f[f_ind].x += g0t.x;
      //f[f_ind].y += g0t.y;
      if( ell0 >= -center_half_size && ell0 < center_half_size &&
          ell1 >= -center_half_size && ell1 < center_half_size )
      {
        /*int f_ind_center = ty % memory_multiplier + 
                           (ell0 + center_half_size) * memory_multiplier +
                           (ell1 + center_half_size) * memory_multiplier *  center_size;

        //f_center[f_ind_center].x += g0t.x;
        //f_center[f_ind_center].y += g0t.y;
        atomicAdd(&(f_center[f_ind_center].x), g0t.x);
        atomicAdd(&(f_center[f_ind_center].y), g0t.y);*/
      } else {
        //f[f_ind].x += g0t.x;
        //f[f_ind].y += g0t.y;
        atomicAdd(&(f[f_ind].x), g0t.x);
        atomicAdd(&(f[f_ind].y), g0t.y);
      }
    }
  }
}

/*m = 4
mu = 2.6356625556996645e-05
n = 362
nproj = 241
nz = 128
g (128, 241, 362)
f (128, 732, 732)
theta (241,)*/

extern "C" __global__ void gather_kernel_center(float2 *g, float2 *f, float *theta, 
                                                int m, float mu,  
                                                int center_size,
                                                int n, int nproj, int nz)            
{

  const int center_half_size = center_size/2;

  //int tx = blockDim.x * blockIdx.x + threadIdx.x;
  //int ty = blockDim.y * blockIdx.y + threadIdx.y;

  int tx = max(0, n + m - center_half_size) + blockDim.x * blockIdx.x + threadIdx.x;
  int ty = max(0, n + m - center_half_size) + blockIdx.y; 
  int tz = blockDim.z * blockIdx.z + threadIdx.z;

  int proj_offset =  threadIdx.y;
  int proj_count  = blockDim.y;

  if (tx >= 2 * n + 2 * m || ty >= 2 * n + 2 * m || tz >= nz)
    return;

  float2 f_value;

  const float coeff0 = M_PI / mu;
  const float coeff1 = -M_PI * M_PI / mu;

  int f_stride = 2*n + 2*m;
  int f_stride_2 = f_stride * f_stride;

  // offset f by [tz, n+m, n+m]
  f += tz * f_stride_2;

  // index of the force
  int f_ind = tx + ty * f_stride;

  float radius_2 = float(m + 1) * float(m + 1) / f_stride_2;

  f_value.x = 0;
  f_value.y = 0;

  // Point coordinates
  float2 point = make_float2(float(tx - (n+m)) / f_stride, float((n+m) - ty) / f_stride);

  for( int proj_index = proj_offset; proj_index < nproj; proj_index+=proj_count) {

    float sintheta, costheta;
    __sincosf(theta[proj_index], &sintheta, &costheta);

    float polar_radius   = 0.5;
    float polar_radius_2 = polar_radius * polar_radius;

    float2 vector_polar = make_float2(polar_radius * costheta, polar_radius * sintheta);
    float2 vector_point = make_float2(point.x,  point.y);

    float dot = vector_polar.x * vector_point.x + vector_polar.y * vector_point.y;
    float2 mid_point = make_float2(dot * vector_polar.x / polar_radius_2, 
                                   dot * vector_polar.y / polar_radius_2); 

    float distance_2 = (mid_point.x - vector_point.x) * (mid_point.x - vector_point.x) +
                       (mid_point.y - vector_point.y) * (mid_point.y - vector_point.y);

    if( radius_2 >= distance_2 ) {
      
      // Distance to intersect
      float distance_to_intersect = sqrtf(radius_2 - distance_2);

      int radius_min, radius_max;
      if( abs(vector_polar.x) > abs(vector_polar.y) ) {
        radius_min = n/2     + floorf((mid_point.x - distance_to_intersect * vector_polar.x / polar_radius) / (2.f * vector_polar.x / n));
        radius_max = n/2 + 1 + floorf((mid_point.x + distance_to_intersect * vector_polar.x / polar_radius) / (2.f * vector_polar.x / n));
      } else {
        radius_min = n/2     + floorf((mid_point.y - distance_to_intersect * vector_polar.y / polar_radius) / (2.f * vector_polar.y / n));
        radius_max = n/2 + 1 + floorf((mid_point.y + distance_to_intersect * vector_polar.y / polar_radius) / (2.f * vector_polar.y / n));
      }

      if( radius_min > radius_max ) {
        int temp(radius_max); radius_max = radius_min; radius_min = temp;
      }

      radius_min = radius_min < 0     ?     0 : radius_min;
      radius_min = radius_min > (n-1) ? (n-1) : radius_min;
      radius_max = radius_max < 0     ?     0 : radius_max;
      radius_max = radius_max > (n-1) ? (n-1) : radius_max;

      for( int radius_index = radius_min; radius_index < radius_max; radius_index++) {

        int g_ind = radius_index + proj_index * n + tz * n * nproj;
 
        float x0 = (radius_index - n / 2) / (float)(n) * costheta;
        float y0 = (radius_index - n / 2) / (float)(n) * sintheta;

        if (x0 >= 0.5f)
          x0 = 0.5f - 1e-5;
        if (y0 >= 0.5f)
          y0 = 0.5f - 1e-5;

        float w0 = point.x - x0;
        float w1 = point.y - y0;
        float w = coeff0 * __expf(coeff1 * (w0 * w0 + w1 * w1));

        float2 g0, g0t;

        g0.x = g[g_ind].x;
        g0.y = g[g_ind].y;
        g0t.x = w*g0.x;
        g0t.y = w*g0.y;

        f_value.x += g0t.x;
        f_value.y += g0t.y;
      }
    }
  }

  f[f_ind].x = f_value.x;
  f[f_ind].y = f_value.y;

  // atomicAdd(&(f[f_ind].x), f_value.x);
  // atomicAdd(&(f[f_ind].y), f_value.y);
}

extern "C" __global__ void wrap_kernel(float2 *f,
                                       int center_size,
                                       int n, int nz, int m)
{
  const int center_half_size = center_size/2;

  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;

  if (tx >= 2 * n + 2 * m || ty >= 2 * n + 2 * m || tz >= nz)
    return;
  if (tx < m || tx >= 2 * n + m || ty < m || ty >= 2 * n + m)
  {
    int tx0 = (tx - m + 2 * n) % (2 * n);
    int ty0 = (ty - m + 2 * n) % (2 * n);
    int id1 = tx + ty * (2 * n + 2 * m) + tz * (2 * n + 2 * m) * (2 * n + 2 * m);
    int id2 = tx0 + m + (ty0 + m) * (2 * n + 2 * m) + tz * (2 * n + 2 * m) * (2 * n + 2 * m);

    atomicAdd(&f[id2].x, f[id1].x);
    atomicAdd(&f[id2].y, f[id1].y);
  } /*else if ( tx >= (n + m - center_half_size) && tx < (n + m + center_half_size) &&
              ty >= (n + m - center_half_size) && ty < (n + m + center_half_size) ) {
  
    int stride1 = 2*n + 2*m;
    int stride2 = stride1 * stride1;

    f += tz * stride2;
    f_center += tz * center_size * center_size;

    int center_index_x = tx - (n + m - center_half_size);
    int center_index_y = ty - (n + m - center_half_size);

    int f_ind = tx + ty * stride1;
    f[f_ind].x = f_center[f_ind].x;
    f[f_ind].y = f_center[f_ind].y;  
  }*/

  
  
  /* else if ( tx >= (n + m - center_half_size) && tx < (n + m + center_half_size) &&
              ty >= (n + m - center_half_size) && ty < (n + m + center_half_size) ) {
    
    int stride1 = 2*n + 2*m;
    int stride2 = stride1 * stride1;
    f += tz * stride2;
    f_center += tz * memory_multiplier * center_size * center_size;

    int center_index_x = tx - (n + m - center_half_size);
    int center_index_y = ty - (n + m - center_half_size);

    f_center += center_index_x * memory_multiplier + center_index_y * memory_multiplier * center_size;

    float2 value = make_float2(0.f, 0.f);
    #pragma unroll
    for(unsigned int i = 0; i < memory_multiplier; i++)
    {
      value.x += f_center[i].x;
      value.y += f_center[i].y;
    }


    int f_ind = tx + ty * stride1;
    atomicAdd(&f[f_ind].x, value.x);
    atomicAdd(&f[f_ind].y, value.y);
  }*/
}
